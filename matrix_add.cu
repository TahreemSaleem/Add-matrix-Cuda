#include "hip/hip_runtime.h"
#include "stdio.h"
__global__
void addSquareMatrix (int *A, int *B, int *result, int n) {
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < n && y < n) {
		result[y * n + x] = A[y * n + x] + B[y * n + x];
		//The same as: result[y][x] = arr1[y][x] + arr2[y][x];
	}
}

int main() {
	hipEvent_t start, stop;

	float t;

	hipEventCreate(&start);

	hipEventRecord(start, 0);
	const int N = 15000;

	int *mat1_h = (int *)malloc(sizeof(int) * N * N);
	int *mat2_h = (int *)malloc(sizeof(int) * N * N);

	int *mat1_d, *mat2_d, *result_d;
	hipMalloc(&mat1_d, sizeof(int) * N * N);
	hipMalloc(&mat2_d, sizeof(int) * N * N);
	hipMalloc(&result_d, sizeof(int) * N * N);

	//cudaMemcpy(mat1_d, mat1_h, sizeof(int) * N * N, cudaMemcpyHostToDevice);
	//cudaMemcpy(mat2_d, mat2_h, sizeof(int) * N * N, cudaMemcpyHostToDevice);
	dim3 dimBlock(256, 256);
	dim3 dimGrid(N/256, N/256);

	addSquareMatrix<<<dimGrid, dimBlock>>>(mat1_d, mat2_d, result_d, N);

	int *result_h = (int *)malloc(sizeof(int) * N);
	//cudaMemcpy(result_h, result_d, sizeof(int) * N, cudaMemcpyDeviceToHost);

	//print results

	hipFree(result_d);
	hipFree(mat1_d);
	hipFree(mat2_d);
	free(mat1_h);
	free(mat2_h);
	free(result_h);
	hipEventCreate(&stop);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop); 

	hipEventElapsedTime(&t, start, stop);
	printf("Time for the kernel: %f ms\n", t);
}
